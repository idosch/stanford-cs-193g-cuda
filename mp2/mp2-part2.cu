#include "hip/hip_runtime.h"
/* This is machine problem 2, part 2: brute force k nearest neighbors
 * You are given a large number of particles, and are asked
 * to find the k particles that are nearest to each one.
 * Look at the example in /tutorials/thread_local_variables.cu
 * for how you can use per thread arrays for sorting.
 * Using that example, port the cpu reference code to the gpu in a first step.
 * In a second step, modify your code so that the per-thread arrays are in 
 * shared memory. You should submit this second version of your code. 
 */
 
#include <cassert>

#include "mp2-util.h"

#define BLOCK_SIZE 256
#define NUM_NEIGH 5

const bool print_debug = false;

event_pair timer;

inline __device__ __host__ float3 operator -(float3 a, float3 b)
{
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__host__ __device__
float dist2(float3 a, float3 b)
{
  float3 d = a - b;
  float d2 = d.x*d.x + d.y*d.y + d.z*d.z;
  return d2;
}

template
<typename T>
__host__ __device__
void init_list(T *base_ptr, unsigned int size, T val)
{
  for(int i=0;i<size;i++)
  {
    base_ptr[i] = val;
  }
}

__host__ __device__
void insert_list(float *dist_list, int *id_list, int size, float dist, int id)
{
 int k;
 for (k=0; k < size; k++) {
   if (dist < dist_list[k]) {
     // we should insert it in here, so push back and make it happen
     for (int j = size - 1; j > k ; j--) {
       dist_list[j] = dist_list[j-1];
       id_list[j] = id_list[j-1];
     }
     dist_list[k] = dist;
     id_list[k] = id;
     break;
   }
 }
}

template
  <int num_neighbors>
void host_find_knn(float3 *particles, int *knn, int array_length)
{
  for(int i=0;i<array_length;i++)
  {
    float3 p = particles[i];
    float neigh_dist[num_neighbors];
    int neigh_ids[num_neighbors];
    
    init_list(&neigh_dist[0],num_neighbors,2.0f);
    init_list(&neigh_ids[0],num_neighbors,-1);
    for(int j=0;j<array_length;j++)
    {
      if(i != j)
      {
        float rsq = dist2(p,particles[j]);
        insert_list(&neigh_dist[0], &neigh_ids[0], num_neighbors, rsq, j);
      }
    }
    for(int j=0;j<num_neighbors;j++)
    {
      knn[num_neighbors*i + j] = neigh_ids[j];
    }
  }
}

__global__ void device_find_knn(float3 *particles, int *knn, int num_particles, int num_neighbors) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_particles) {
    float3 p = particles[i];  // the particle whose neighbors we wish to find
    float neigh_dist[NUM_NEIGH];  // distances of the k neighbors from p
    int neigh_ids[NUM_NEIGH]; // their ids

    init_list(&neigh_dist[0], num_neighbors, 2.0f); // initialize distances
    init_list(&neigh_ids[0], num_neighbors, -1);  // initialize ids

    // iterate over all the other particles and find nearest neighbors
    for (int j = 0; j < num_particles; j++) {
      if (i != j) {
        float rsq = dist2(p, particles[j]);
        insert_list(&neigh_dist[0], &neigh_ids[0], num_neighbors, rsq, j);
      }
    }

    // update the global nearest neighbors matrix
    // please note that no atomic functions are needed as each thread accesses
    // different elements in this global matrix
    for (int j = 0; j < num_neighbors; j++) {
      knn[i*num_neighbors + j] = neigh_ids[j];
    }
 }
}

__global__ void device_find_knn_smem(float3 *particles, int *knn, int num_particles, int num_neighbors) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_particles) {
    float3 p = particles[i];  // the particle whose neighbors we wish to find
    __shared__ float neigh_dist[BLOCK_SIZE*NUM_NEIGH];  // distances of the k neighbors from p
    __shared__ int neigh_ids[BLOCK_SIZE*NUM_NEIGH]; // their ids

    init_list(&neigh_dist[threadIdx.x*num_neighbors], num_neighbors, 2.0f); // initialize distances
    init_list(&neigh_ids[threadIdx.x*num_neighbors], num_neighbors, -1);  // initialize ids

    // iterate over all the other particles and find nearest neighbors
    for (int j = 0; j < num_particles; j++) {
      if (i != j) {
        float rsq = dist2(p, particles[j]);
        insert_list(&neigh_dist[threadIdx.x*num_neighbors], &neigh_ids[threadIdx.x*num_neighbors], num_neighbors, rsq, j);
      }
    }

    // update the global nearest neighbors matrix
    // please note that no atomic functions are needed as each thread accesses
    // different elements in this global matrix
    for (int j = 0; j < num_neighbors; j++) {
      knn[i*num_neighbors + j] = neigh_ids[threadIdx.x*num_neighbors + j];
    }
 }
}

void allocate_host_memory(int num_particles, int num_neighbors,
                          float3 *&h_particles, int *&h_knn, int *&h_knn_checker)
{
  // malloc host array
  h_particles = (float3*)malloc(num_particles * sizeof(float3));
  h_knn = (int*)malloc(num_particles * num_neighbors * sizeof(int));
  h_knn_checker = (int*)malloc(num_particles * num_neighbors * sizeof(int));

  // if either memory allocation failed, report an error message
  if(h_particles == 0 || h_knn == 0 || h_knn_checker == 0)
  {
    printf("couldn't allocate host memory\n");
    exit(1);
  }
}


void allocate_device_memory(int num_particles, int num_neighbors,
                            float3* &d_particles, int* &d_knn) {

  hipMalloc((void**)&d_particles, num_particles * sizeof(float3));
  hipMalloc((void**)&d_knn, num_particles * num_neighbors * sizeof(int));
}


void deallocate_host_memory(float3 *h_particles, int *h_knn, int *h_knn_checker)
{
  free(h_particles);
  free(h_knn);
  free(h_knn_checker);
}


void deallocate_device_memory(float3 *d_particles, int *d_knn) {
  hipFree(d_particles);
  hipFree(d_knn);
}


bool cross_check_results(int * reference_knn, int * knn, int num_particles, int num_neighbors)
{
  int error = 0;


  for(int i=0;i<num_particles;i++)
  {
    for(int j=0;j<num_neighbors;j++)
    {
      if(reference_knn[i*num_neighbors + j] != knn[i*num_neighbors + j])
      {
        if(print_debug) printf("particle %d, neighbor %d is %d on cpu, %d on gpu\n",i,j,reference_knn[i*num_neighbors + j],knn[i*num_neighbors + j]);
        error = 1;
      }
    }

  }

  if(error)
  {
    printf("Output of CUDA version and normal version didn't match! \n");
  }
  else {
    printf("Worked! CUDA and reference output match. \n");
  }
  return error;
}

int main(void)
{  
  // create arrays of 8K elements
  int num_particles = 20*1024;
  const int num_neighbors = 5;

  // pointers to host arrays
  float3 *h_particles = 0;
  int    *h_knn = 0;
  int    *h_knn_checker = 0;

  // pointers to device arrays
  float3 *d_particles = 0;
  int    *d_knn = 0;

  allocate_host_memory(num_particles, num_neighbors, h_particles, h_knn, h_knn_checker);
  allocate_device_memory(num_particles, num_neighbors, d_particles, d_knn);

  // generate random input
  // initialize
  srand(13);

  for(int i=0;i< num_particles;i++)
  {
    h_particles[i] = make_float3((float)rand()/(float)RAND_MAX,(float)rand()/(float)RAND_MAX,(float)rand()/(float)RAND_MAX);
  }

  // copy input to GPU
  start_timer(&timer);
  hipMemcpy(d_particles, h_particles, num_particles * sizeof(float3), hipMemcpyHostToDevice);
  stop_timer(&timer,"copy to gpu");

  start_timer(&timer);
  device_find_knn<<<num_particles / BLOCK_SIZE, BLOCK_SIZE >>>(d_particles, d_knn, num_particles, num_neighbors);
  check_cuda_error("brute force knn");
  stop_timer(&timer,"brute force knn");

  start_timer(&timer);  
  device_find_knn_smem<<<num_particles / BLOCK_SIZE, BLOCK_SIZE >>>(d_particles, d_knn, num_particles, num_neighbors);
  check_cuda_error("shared meme knn");
  stop_timer(&timer,"shared mem knn");

  // download and inspect the result on the host
  start_timer(&timer);
  hipMemcpy(h_knn, d_knn, num_particles * num_neighbors * sizeof(int), hipMemcpyDeviceToHost);
  check_cuda_error("copy from gpu");
  stop_timer(&timer,"copy back from gpu memory");

  // generate reference output
  start_timer(&timer);
  host_find_knn<num_neighbors>(h_particles, h_knn_checker, num_particles);
  stop_timer(&timer,"cpu brute force knn");

  // check CUDA output versus reference output
  cross_check_results(h_knn_checker, h_knn, num_particles, num_neighbors);

  deallocate_host_memory(h_particles, h_knn, h_knn_checker);
  deallocate_device_memory(d_particles, d_knn);

  return 0;
}

